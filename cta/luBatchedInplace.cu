#include "luBatchedInplace.cuh"

int main() {
    int matrixSize, numMatrices, numThreads;

    std::cout << "Enter matrix size: ";
    std::cin >> matrixSize;
    std::cout << "Enter number of matrices: ";
    std::cin >> numMatrices;
    std::cout << "Enter number of threads in a block: ";
    std::cin >> numThreads;
    // numThreads = 32;
    int threadsPerMatrix = matrixSize;
    int matricesPerBlock = numThreads / threadsPerMatrix;
    int numBlocks = numMatrices / matricesPerBlock;

    int numElements = matrixSize * matrixSize * numMatrices;

    std::vector<FpType> A(numElements);
    std::vector<FpType> A_inv(numElements);

    FpType* d_A;
    CUDA_CHECK(hipMallocManaged(&d_A, numElements * sizeof(FpType)));

    FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};

    for (int k = 0; k < numMatrices; ++k) {
        int offset = k * matrixSize * matrixSize;
        std::ifstream file("matrix.txt");
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                file >> A[(i * matrixSize) + offset + j];
                // A[(i * matrixSize) + offset + j] = inputMatrix[(i * matrixSize) + j];
                // A[(i * matrixSize) + offset + j] = rand() % 10;
                A_inv[(i * matrixSize) + offset + j] = static_cast<FpType>(0.0);
            }
        }
        file.close();
    }

    // printMatrices(A, matrixSize, numMatrices);

    CUDA_CHECK(hipMemcpy(d_A, A.data(), numElements * sizeof(FpType), hipMemcpyHostToDevice));
    std::cout << "Data copied to device." << '\n';

    int shMemSize = matricesPerBlock * matrixSize * matrixSize * sizeof(FpType);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(batched_lu_subwarp<FpType>), hipFuncAttributeMaxDynamicSharedMemorySize, shMemSize));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    batched_lu_subwarp<FpType><<<numBlocks, numThreads, shMemSize>>>(d_A, matrixSize, numMatrices, threadsPerMatrix, matricesPerBlock);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    CUDA_CHECK(hipMemcpy(A_inv.data(), d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost));
    std::cout << "Data copied back to host." << '\n';

    // print A_inv
    // printMatrices(A_inv, matrixSize, numMatrices);

    auto startT = std::chrono::high_resolution_clock::now();
    verifyInv(A, A_inv, matrixSize, numMatrices);
    auto endT = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = endT - startT;
    std::cout << "Time taken to verify inverse: " << elapsed.count() << " seconds\n";
    
    hipFree(d_A);

    return 0;
}
